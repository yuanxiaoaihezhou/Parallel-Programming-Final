#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"

// 定义矩阵类型
using matrix = std::vector<std::vector<float>>;

// CUDA 核函数，用于并行处理每一行的消元
__global__ void elimination_kernel(float* m_dev, int n, int i, float* pivot_row) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < n && k != i) {
        float factor = m_dev[k * (n + 1) + i];
        for (int j = i; j < n + 1; ++j) {
            m_dev[k * (n + 1) + j] -= factor * pivot_row[j];
        }
    }
}

// CUDA 主函数
void gaussian_elimination_cuda(matrix& m) {
    int n = m.size();
    int matrix_elements = n * (n + 1);
    size_t matrix_size_bytes = matrix_elements * sizeof(float);
    size_t row_size_bytes = (n + 1) * sizeof(float);

    // 1. 在GPU上分配内存
    float* m_dev;
    hipMalloc(&m_dev, matrix_size_bytes);

    // 将二维vector转为一维数组以便传输
    std::vector<float> m_flat;
    m_flat.reserve(matrix_elements);
    for(const auto& row : m) {
        m_flat.insert(m_flat.end(), row.begin(), row.end());
    }

    // 2. 将数据从CPU拷贝到GPU
    hipMemcpy(m_dev, m_flat.data(), matrix_size_bytes, hipMemcpyHostToDevice);

    // 为主元行在GPU上分配内存
    float* pivot_row_dev;
    hipMalloc(&pivot_row_dev, row_size_bytes);
    
    // 3. 高斯消去主循环 (在CPU中控制)
    for (int i = 0; i < n; ++i) {
        // (寻找主元在GPU上做很低效，通常在CPU上完成)
        // 这里为了简化，我们假设主元就是m[i][i]且不为0
        // 一个完整的实现需要将列数据拷回CPU找主元，再交换行指针
        
        // 将主元行拷贝到GPU上的专用内存
        hipMemcpy(pivot_row_dev, m_dev + i * (n + 1), row_size_bytes, hipMemcpyDeviceToDevice);
        
        // 归一化主元行 (可以在一个简单的kernel里做，或者就在CPU上算完再传)
        // 这里为了简化，我们依然在CPU上操作，然后把更新后的主元行传回去
        // 这不是最高效的，但能展示基本思路
        std::vector<float> temp_row(n + 1);
        hipMemcpy(temp_row.data(), pivot_row_dev, row_size_bytes, hipMemcpyDeviceToHost);
        float pivot_val = temp_row[i];
        for(int j=i; j<n+1; ++j) {
            temp_row[j] /= pivot_val;
        }
        hipMemcpy(pivot_row_dev, temp_row.data(), row_size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(m_dev + i * (n+1), temp_row.data(), row_size_bytes, hipMemcpyHostToDevice);
        

        // 设置kernel启动参数
        int threads_per_block = 256;
        int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

        // 4. 调用kernel执行并行消元
        elimination_kernel<<<blocks_per_grid, threads_per_block>>>(m_dev, n, i, pivot_row_dev);

        // 同步以确保kernel执行完毕
        hipDeviceSynchronize();
    }
    
    // 5. 将结果从GPU拷回CPU
    hipMemcpy(m_flat.data(), m_dev, matrix_size_bytes, hipMemcpyDeviceToHost);

    // 6. 释放GPU内存
    hipFree(m_dev);
    hipFree(pivot_row_dev);

    // 将一维数组结果转回二维vector
    for(int i = 0; i < n; ++i) {
        for(int j = 0; j < n + 1; ++j) {
            m[i][j] = m_flat[i * (n + 1) + j];
        }
    }
}

// C++ Wrapper, 这是 main.cpp 会调用的函数
void gaussian_elimination_cuda_wrapper(matrix& m) {
    gaussian_elimination_cuda(m);
}